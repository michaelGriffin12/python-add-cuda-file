
#include <hip/hip_runtime.h>
__global__ void
do_add(int *ar1, int *ar2, int *out)
{
  int ctid;
  int cwd;

  ctid = threadIdx.x + (blockIdx.x * blockDim.x);

  cwd = ar1[ctid];
  cwd = ar2[ctid] + cwd;
  out[ctid] = cwd;

  return;
}

void
load_kernel(int *ar1, int *ar2, int *out)
{
  do_add<<<1, 32>>>(ar1, ar2, out);
  return;
}
